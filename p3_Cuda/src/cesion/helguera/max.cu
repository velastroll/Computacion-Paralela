#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <stdlib.h>
#include <ctime>
#include "kernel.cu"



int main()
{
	unsigned int N = 1024*1024*20;
	float *h_array;
	float *d_array;
	float *h_max;
	float *d_max;
	int *d_mutex;


	// allocate memory
	h_array = (float*)malloc(N*sizeof(float));
	h_max = (float*)malloc(sizeof(float));
	hipMalloc((void**)&d_array, N*sizeof(float));
	hipMalloc((void**)&d_max, sizeof(float));
	hipMalloc((void**)&d_mutex, sizeof(int));
	hipMemset(d_max, 0, sizeof(float));
	hipMemset(d_mutex, 0, sizeof(float));


	// fill host array with data
	for(unsigned int i=0;i<N;i++){
		h_array[i] = i;
	}


	// set up timing variables
	float gpu_elapsed_time;
	hipEvent_t gpu_start, gpu_stop;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);


	// copy from host to device
	hipEventRecord(gpu_start, 0);
	hipMemcpy(d_array, h_array, N*sizeof(float), hipMemcpyHostToDevice);


	// call kernel
	
	dim3 block(1,1204);
	dim3 grid(1,1);
	find_maximum_kernel<<< grid, block >>>(d_array, d_max, d_mutex, N);
	


	// copy from device to host
	hipMemcpy(h_max, d_max, sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);


	//report results
	std::cout<<"Maximo gpu: "<<*h_max<<std::endl;
	//std::cout<<"The gpu took: "<<gpu_elapsed_time<<" milli-seconds"<<std::endl;


	/*//run cpu version
	clock_t cpu_start = clock();
	//for(unsigned int j=0;j<1000;j++){
		*h_max = -1.0;
		for(unsigned int i=0;i<N;i++){
			if(h_array[i] > *h_max){
				*h_max = h_array[i];
			}
		}
	//}
	clock_t cpu_stop = clock();
	clock_t cpu_elapsed_time = 1000*(cpu_stop - cpu_start)/CLOCKS_PER_SEC;

	std::cout<<"Maximo: "<<*h_max<<std::endl;
	std::cout<<"CPU: "<<cpu_elapsed_time<<" milli-seconds"<<std::endl;*/



	// free memory
	free(h_array);
	free(h_max);
	hipFree(d_array);
	hipFree(d_max);
    hipFree(d_mutex);
    
    return 0;
}