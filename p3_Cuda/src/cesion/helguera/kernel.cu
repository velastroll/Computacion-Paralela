
#include <hip/hip_runtime.h>
__global__ void ReductionMax2(float *input, float *results, int n)    //take thread divergence into account
{
	extern __shared__ int sdata[];
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tx = threadIdx.x;
	 //load input into __shared__ memory
	int x = INT_MIN;
	if(i < n)
		x = input[i];
	sdata[tx] = x;
	__syncthreads();

	// block-wide reduction
	for(unsigned int offset = blockDim.x>>1; offset > 0; offset >>= 1)
	{
		__syncthreads();
		if(tx < offset)
	    {
			if(sdata[tx + offset] > sdata[tx])
				sdata[tx] = sdata[tx + offset];
		}

	}

		// finally, thread 0 writes the result
	if(threadIdx.x == 0)
	{
		// the result is per-block
		results[blockIdx.x] = sdata[0];
	}
}

__global__ void find_maximum_kernel(float *array, float *max, int *mutex, unsigned int n)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	__shared__ float cache[256];


	float temp = -1.0;
	while(index + offset < n){
		temp = fmaxf(temp, array[index + offset]);

		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();


	// reduction
	unsigned int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	if(threadIdx.x == 0){
		while(atomicCAS(mutex,0,1) != 0);  //lock
		*max = fmaxf(*max, cache[0]);
		atomicExch(mutex, 0);  //unlock
	}
}