/*
 * Simulacion simplificada de bombardeo de particulas de alta energia
 *
 * Computacion Paralela (Grado en Informatica)
 * 2017/2018
 *
 * (c) 2018 Arturo Gonzalez Escribano
 */
 #include<stdio.h>
 #include<stdlib.h>
 #include<math.h>
 #include<hip/hip_runtime.h>
 #include"cputils.h"
 #include"kernel.cu"
 
 #define PI	3.14159f
 #define HILOS 128
 
 /* Estructura para almacenar los datos de una tormenta de particulas */
 typedef struct {
	 int size;
	 int *posval;
 } Storm;
 
 
 /* FUNCIONES AUXILIARES: No se utilizan dentro de la medida de tiempo, dejar como estan */
 /* Funcion de DEBUG: Imprimir el estado de la capa */
 void debug_print(int layer_size, float *layer, int *posiciones, float *maximos, int num_storms ) {
	 int i,k;
	 if ( layer_size <= 35 ) {
		 /* Recorrer capa */
		 for( k=0; k<layer_size; k++ ) {
			 /* Escribir valor del punto */
			 printf("%10.4f |", layer[k] );
 
			 /* Calcular el numero de caracteres normalizado con el maximo a 60 */
			 int ticks = (int)( 60 * layer[k] / maximos[num_storms-1] );
 
			 /* Escribir todos los caracteres menos el ultimo */
			 for (i=0; i<ticks-1; i++ ) printf("o");
 
			 /* Para maximos locales escribir ultimo caracter especial */
			 if ( k>0 && k<layer_size-1 && layer[k] > layer[k-1] && layer[k] > layer[k+1] )
				 printf("x");
			 else
				 printf("o");
 
			 /* Si el punto es uno de los maximos especiales, annadir marca */
			 for (i=0; i<num_storms; i++)
				 if ( posiciones[i] == k ) printf(" M%d", i );
 
			 /* Fin de linea */
			 printf("\n");
		 }
	 }
 }
 
 /*
  * Funcion: Lectura de fichero con datos de tormenta de particulas
  */
 Storm read_storm_file( char *fname ) {
	 FILE *fstorm = cp_abrir_fichero( fname );
	 if ( fstorm == NULL ) {
		 fprintf(stderr,"Error: Opening storm file %s\n", fname );
		 exit( EXIT_FAILURE );
	 }
 
	 Storm storm;
	 int ok = fscanf(fstorm, "%d", &(storm.size) );
	 if ( ok != 1 ) {
		 fprintf(stderr,"Error: Reading size of storm file %s\n", fname );
		 exit( EXIT_FAILURE );
	 }
 
	 storm.posval = (int *)malloc( sizeof(int) * storm.size * 2 );
	 if ( storm.posval == NULL ) {
		 fprintf(stderr,"Error: Allocating memory for storm file %s, with size %d\n", fname, storm.size );
		 exit( EXIT_FAILURE );
	 }
 
	 int elem;
	 for ( elem=0; elem<storm.size; elem++ ) {
		 ok = fscanf(fstorm, "%d %d\n",
					 &(storm.posval[elem*2]),
					 &(storm.posval[elem*2+1]) );
		 if ( ok != 2 ) {
			 fprintf(stderr,"Error: Reading element %d in storm file %s\n", elem, fname );
			 exit( EXIT_FAILURE );
		 }
	 }
	 fclose( fstorm );
 
	 return storm;
 }
 
 
 /*
  * PROGRAMA PRINCIPAL
  */
 int main(int argc, char *argv[]) {
	 int i,j,k;
 
	 /* 1.1. Leer argumentos */
	 if (argc<3) {
		 fprintf(stderr,"Usage: %s <size> <storm_1_file> [ <storm_i_file> ] ... \n", argv[0] );
		 exit( EXIT_FAILURE );
	 }
 
	 int layer_size = atoi( argv[1] );
	 int num_storms = argc-2;
	 Storm storms[ num_storms ];
 
	 /* 1.2. Leer datos de storms */
	 for( i=2; i<argc; i++ )
		 storms[i-2] = read_storm_file( argv[i] );
 
	 /* 1.3. Inicializar maximos a cero */
	 float maximos[ num_storms ];
	 int posiciones[ num_storms ];
	 for (i=0; i<num_storms; i++) {
		 maximos[i] = 0.0f;
		 posiciones[i] = 0;
	 }
 
	 /* 2. Inicia medida de tiempo */
	 hipSetDevice(0);
	 hipDeviceSynchronize();
	 double ttotal = cp_Wtime();
 
	 /* COMIENZO: No optimizar/paralelizar el main por encima de este punto */

	 float *layer;
	 hipMalloc(&layer, sizeof(float)*layer_size);
	 float *layer_copy;
	 hipMalloc(&layer_copy, sizeof(float)*layer_size);

	 if ( layer == NULL || layer_copy == NULL ) {
		 fprintf(stderr,"Error: Allocating the layer memory\n");
		 exit( EXIT_FAILURE );
	 }
	 
	for( k=0; k<layer_size; k++ ) layer[k] = 0.0f;
	for( k=0; k<layer_size; k++ ) layer_copy[k] = 0.0f;

	hipError_t err;

	float *d_layer;
	float *d_layerCopy;
	float *h_max;
	int *h_pos;

	hipMalloc(&h_max, sizeof(float)*layer_size);
	hipMalloc(&h_pos, sizeof(float)*layer_size);

	err = hipMalloc((void**) &d_layer,layer_size*sizeof(float));
	if (err != hipSuccess) printf("CUDA-ERROR 1: %s\n", hipGetErrorString(err));
	err = hipMalloc((void**) &d_layerCopy,layer_size*sizeof(float));
	if (err != hipSuccess) printf("CUDA-ERROR 2: %s\n", hipGetErrorString(err));
	
	err = hipMemcpy(d_layer, layer, layer_size * sizeof(float), hipMemcpyHostToDevice );
	if (err != hipSuccess) printf("CUDA-ERROR 4: %s\n", hipGetErrorString(err));
	err = hipMemcpy(d_layerCopy, layer_copy, layer_size * sizeof(float), hipMemcpyHostToDevice );
	if (err != hipSuccess) printf("CUDA-ERROR 5: %s\n", hipGetErrorString(err));
	
	
	float p;
	int BLOQUES = (1+(layer_size-1)/HILOS);
	if (BLOQUES<1) BLOQUES=1;

	for( i=0; i<num_storms; i++) {
		
		for (p=0; p<((layer_size));p++){
			h_pos[p]=p;
		}

		err = hipMemcpy(d_layerCopy, h_pos, ((layer_size))* sizeof(float), hipMemcpyHostToDevice );
		if (err != hipSuccess) printf("CUDA-ERROR 6: %s\n", hipGetErrorString(err));

		for( j=0; j<storms[i].size; j++ ) {
			float energia = (float)storms[i].posval[j*2+1] / 1000;
			int posicion = storms[i].posval[j*2];
			gpu_Actualizar<<<BLOQUES, HILOS>>>(d_layer, posicion, energia,layer_size);
		}
		
		gpu_Copiar<<<BLOQUES, HILOS>>>(d_layer, d_layerCopy,layer_size);
		
		gpu_Relajacion<<<BLOQUES, HILOS>>>(d_layer, d_layerCopy, layer_size);


		gpu_Copiar<<<BLOQUES, HILOS>>>(d_layer, d_layerCopy, layer_size);

		// TENEMOS LOS DATOS EN D_LAYER Y EN D_LAYERCOPY
		// HACEMOS LA COPIA EN LAYER
		err = hipMemcpy(layer, d_layer, sizeof(float)*layer_size, hipMemcpyDeviceToHost );
		if (err != hipSuccess) printf("CUDA-ERROR 8: %s\n", hipGetErrorString(err));
		// Usamos D_LAYERCOPY como POSICIONES
		// Metemos los condidatos en D_LAYER

		gpu_obtenCandidatos<<<BLOQUES, HILOS>>>(d_layerCopy, d_layer, layer_size);

		for (int redSize = layer_size; redSize>=1; redSize /= 2) {
			// Usamos LayerCopy como Posiciones
			gpu_reduceMaximo<<<BLOQUES, HILOS>>> (d_layer, d_layerCopy, redSize);
		}

		err = hipMemcpy(h_pos, d_layerCopy, sizeof(float)*((layer_size)), hipMemcpyDeviceToHost );
		if (err != hipSuccess) printf("CUDA-ERROR 8: %s\n", hipGetErrorString(err));

		err = hipMemcpy(h_max, d_layer, sizeof(float)*layer_size, hipMemcpyDeviceToHost );
		if (err != hipSuccess) printf("CUDA-ERROR 8: %s\n", hipGetErrorString(err));

		maximos[i] = h_max[0];
		posiciones[i] = h_pos[0];

		err = hipMemcpy(d_layer, layer, sizeof(float)*layer_size, hipMemcpyHostToDevice );
		if (err != hipSuccess) printf("CUDA-ERROR 8: %s\n", hipGetErrorString(err));


	 }

	 //Liberamos memoria
	err = hipFree(d_layer);
	if (err != hipSuccess) printf("CUDA-ERROR 9: %s\n",hipGetErrorString(err));
	err = hipFree(d_layerCopy);
	if (err != hipSuccess) printf("CUDA-ERROR 10: %s\n", hipGetErrorString(err));


	 /* FINAL: No optimizar/paralelizar por debajo de este punto */
 
	 /* 6. Final de medida de tiempo */
	 ttotal = cp_Wtime() - ttotal;
 
	 /* 7. DEBUG: Dibujar resultado (Solo para capas con hasta 35 puntos) */
	 #ifdef DEBUG
	 debug_print( layer_size, layer, posiciones, maximos, num_storms );
	 #endif
 
	 /* 8. Salida de resultados para tablon */
	 printf("\n");
	 /* 8.1. Tiempo total de la computacion */
	 printf("Time: %lf\n", ttotal );
	 /* 8.2. Escribir los maximos */
	 printf("Result:");
	 for (i=0; i<num_storms; i++)
		 printf(" %d %f", posiciones[i], maximos[i] );
	 printf("\n");
 
	 /* 9. Liberar recursos */
	 for( i=0; i<argc-2; i++ )
		 free( storms[i].posval );
 
	 /* 10. Final correcto */
	 return 0;
 }
 